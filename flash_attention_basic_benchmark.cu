#include <iostream>
#include <vector>
#include <cmath>
#include <cfloat>
#include <cassert>
#include <chrono>
#include <iomanip>
#include <algorithm>  // for min_element, max_element
#include <numeric>    // for accumulate
#include <hip/hip_runtime.h>
#include <hipblas.h>

// CUDA check macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// cuBLAS check macro
#define CUBLAS_CHECK(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS error in " << __FILE__ << ":" << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

void flash_attention_cpu(const float* Q, const float* K, const float* V, float* output, int B, int N, int H, int d) {
    std::vector<float> scores(N * N);
    bool debug = false;

    for (int b = 0; b < B; ++b) {
        for (int h = 0; h < H; ++h) {
            const float* q = Q + b * H * N * d + h * N * d;
            const float* k = K + b * H * N * d + h * N * d;
            const float* v = V + b * H * N * d + h * N * d;
            float* o = output + b * H * N * d + h * N * d;

            // Compute Q * K^T (row-major)
            for (int i = 0; i < N; ++i) {
                for (int j = 0; j < N; ++j) {
                    float sum = 0.0f;
                    for (int d_i = 0; d_i < d; ++d_i) {
                        // Q[i,d_i] * K[j,d_i] for row-major layout
                        sum += q[i * d + d_i] * k[j * d + d_i];
                    }
                    scores[i * N + j] = sum / sqrtf(d);
                    
                }
            }

            if (debug) {
                //print scores
                std::cout << "CPU Scores: ";
                for (int i = 0; i < N * N; ++i) {
                    std::cout << scores[i] << " ";
                }
                std::cout << std::endl;
            }

            // Apply softmax row by row
            for (int i = 0; i < N; ++i) {
                // first find max for numerical stability
                float max_val = -std::numeric_limits<float>::infinity();
                for (int j = 0; j < N; ++j) {
                    max_val = std::max(max_val, scores[i * N + j]);
                }

                // compute exp and sum
                float sum_exp = 0.0f;
                for (int j = 0; j < N; ++j) {
                    scores[i * N + j] = expf(scores[i * N + j] - max_val);
                    sum_exp += scores[i * N + j];
                }

                // normalize
                for (int j = 0; j < N; ++j) {
                    scores[i * N + j] /= sum_exp;
                }
            }

            if (debug) {
                // print cpu softmax
                std::cout << "CPU Softmax: ";
                for (int i = 0; i < N * N; ++i) {
                    std::cout << scores[i] << " ";
                }
                std::cout << std::endl;
            }

            // Compute scores * V (row-major)
            for (int i = 0; i < N; ++i) {
                for (int d_i = 0; d_i < d; ++d_i) {
                    float sum = 0.0f;
                    for (int j = 0; j < N; ++j) {
                        // scores[i,j] * V[j,d_i] for row-major layout
                        sum += scores[i * N + j] * v[j * d + d_i];
                    }
                    o[i * d + d_i] = sum;
                }
            }
        }
    }
}

void initialize_tensor(std::vector<float>& tensor, int size) {
    for (int i = 0; i < size; ++i) {
        tensor[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

struct BenchmarkResults {
    double avg_time_ms;
    double min_time_ms;
    double max_time_ms;
    std::vector<double> individual_times;
};

BenchmarkResults run_cpu_benchmark(const std::vector<float>& h_Q, 
                                 const std::vector<float>& h_K,
                                 const std::vector<float>& h_V,
                                 std::vector<float>& h_output,
                                 int B, int N, int H, int d,
                                 int num_iterations) {
    BenchmarkResults results;
    results.individual_times.reserve(num_iterations);

    for (int i = 0; i < num_iterations; ++i) {
        auto start = std::chrono::high_resolution_clock::now();
        
        flash_attention_cpu(h_Q.data(), h_K.data(), h_V.data(), h_output.data(), B, N, H, d);
        
        auto end = std::chrono::high_resolution_clock::now();
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count();
        results.individual_times.push_back(time_ms);
    }

    // Calculate statistics
    results.min_time_ms = *std::min_element(results.individual_times.begin(), results.individual_times.end());
    results.max_time_ms = *std::max_element(results.individual_times.begin(), results.individual_times.end());
    results.avg_time_ms = std::accumulate(results.individual_times.begin(), results.individual_times.end(), 0.0) / num_iterations;

    return results;
}

__global__ void columnwise_softmax(float* input, float* output, int N) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread handles one column
    if (tid < N) {
        // Find max element in the column for numerical stability
        float max_val = input[tid];
        for (int i = 1; i < N; i++) {
            max_val = fmaxf(max_val, input[i * N + tid]);
        }
        
        // Calculate exp(x - max) and sum
        float sum = 0.0f;
        for (int i = 0; i < N; i++) {
            float val = expf(input[i * N + tid] - max_val);
            output[i * N + tid] = val;
            sum += val;
        }
        
        // Normalize by sum
        for (int i = 0; i < N; i++) {
            output[i * N + tid] /= sum;
        }
    }
}

// V is specifically in column-major format
BenchmarkResults run_gpu_benchmark(hipblasHandle_t handle,
                                 float* d_Q, float* d_K, float* d_V, float* d_output,
                                 int B, int N, int H, int d,
                                 int num_iterations) {
    BenchmarkResults results;
    bool debug = false;
    results.individual_times.reserve(num_iterations);

    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
    CUBLAS_CHECK(hipblasSetStream(handle, 0));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Calculate leading dimensions and strides
    int ldq = d;      
    int ldk = d;      
    int lds = N;    

    for (int i = 0; i < num_iterations; ++i) {
        CUDA_CHECK(hipEventRecord(start));

        float alpha_qk = 1.0f / sqrtf(d);
        float alpha_v = 1.0f;
        float beta = 0.0f;

        float* d_scores;
        CUDA_CHECK(hipMalloc(&d_scores, B * H * N * N * sizeof(float)));

        // Process each batch and head
        for (int b = 0; b < B; ++b) {
            for (int h = 0; h < H; ++h) {
                float* current_Q = d_Q + (b * H * N * d) + (h * N * d);
                float* current_K = d_K + (b * H * N * d) + (h * N * d);
                float* current_V = d_V + (b * H * N * d) + (h * N * d);
                float* current_scores = d_scores + (b * H * N * N) + (h * N * N);
                float* current_output = d_output + (b * H * N * d) + (h * N * d);

                // Q * K^T computation
                CUBLAS_CHECK(hipblasSgemm(handle,
                    HIPBLAS_OP_T,  
                    HIPBLAS_OP_N, 
                    N, N, d,        // m, n, k dimensions
                    &alpha_qk,      // scaling factor
                    current_Q, ldq,  // Q matrix, N x d
                    current_K, ldk,  // K matrix, N x d
                    &beta,   
                    current_scores, lds)); // output scores which is N x N

                if (debug) {
                    // print current scores
                    float* h_scores = new float[N * N];
                    CUDA_CHECK(hipMemcpy(h_scores, current_scores, N * N * sizeof(float), hipMemcpyDeviceToHost));
                    std::cout << "Scores: ";
                    for (int i = 0; i < N * N; ++i) {
                        std::cout << h_scores[i] << " ";
                    }
                    std::cout << std::endl;
                }

                int threadsPerBlock = std::min(N, 1024);
                int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
                columnwise_softmax<<<blocksPerGrid, threadsPerBlock>>>(current_scores, current_scores, N);
                CUDA_CHECK(hipGetLastError());

                if (debug) {
                    // print gpu softmax
                    float* h_scores_softmax = new float[N * N];
                    CUDA_CHECK(hipMemcpy(h_scores_softmax, current_scores, N * N * sizeof(float), hipMemcpyDeviceToHost));
                    std::cout << "GPU Softmax: ";
                    for (int i = 0; i < N * N; ++i) {
                        std::cout << h_scores_softmax[i] << " ";
                    }
                    std::cout << std::endl;
                }
                
                CUBLAS_CHECK(hipblasSgemm(handle,
                    HIPBLAS_OP_N,  
                    HIPBLAS_OP_N, 
                    N, d, N,        // m, n, k dimensions
                    &alpha_v,       // same scale
                    current_scores, lds, // scores matrix which are N x N
                    current_V, N,     // V matrix is N x d
                    &beta,    
                    current_output, N)); // output matrix is N x d
                
            }
        }

        CUDA_CHECK(hipFree(d_scores));
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));

        float time_ms;
        CUDA_CHECK(hipEventElapsedTime(&time_ms, start, stop));
        results.individual_times.push_back(time_ms);
    }

    std::sort(results.individual_times.begin(), results.individual_times.end());
    results.min_time_ms = results.individual_times.front();
    results.max_time_ms = results.individual_times.back();
    results.avg_time_ms = std::accumulate(results.individual_times.begin(), results.individual_times.end(), 0.0) / num_iterations;
    //results.median_time_ms = results.individual_times[num_iterations / 2];

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return results;
}

__global__
void flash_attention_kernel(
    const float* Q,   // [B, H, N, D]
    const float* K,     // [B, H, N, D]
    const float* V,      // [B, H, N, D]
    const int N,       // sequence length
    const int d,   // hidden dimension
    const int num_col_tiles,    
    const int num_row_tiles,   
    const int col_tile_size,   
    const int row_tile_size,   
    const float scale,
    float* l,      // running sum [B, H, N]
    float* m,      // running max [B, H, N]
    float* O       // output [B, H, N, D]
) {
    int threadId = threadIdx.x;
    int batch_idx = blockIdx.x; int head_idx = blockIdx.y;

    int qkv_offset = (batch_idx * gridDim.y * N * d) + (head_idx * N * d);
    int l_m_offset = (batch_idx * gridDim.y * N) + (head_idx * N);

    extern __shared__ float shmem[];
    int tile_size = col_tile_size * d;
    float* Qi = shmem;
    float* Kj = &shmem[tile_size];
    float* Vj = &shmem[tile_size * 2];
    float* S = &shmem[tile_size * 3];

    for (int j = 0; j < num_col_tiles; j++) {
        // Load in Kj, Vj to shmem
        for (int x = 0; x < d; x++) {
            int shmem_idx = (threadId * d) + x;
            int idx = qkv_offset + (tile_size * j) + shmem_idx;
            Kj[shmem_idx] = K[idx];
            Vj[shmem_idx] = V[idx];
        }
        __syncthreads();

        for (int i = 0; i < num_row_tiles; i++) {
            // Load Qi to shmem
            // Load l and m to registers
            for (int x = 0; x < d; x++) {
                Qi[(threadId * d) + x] = Q[qkv_offset + (tile_size * i) + (threadId * d) + x];
            }
            float row_m_prev = m[l_m_offset + (row_tile_size * i) + threadId];
            float row_l_prev = l[l_m_offset + (row_tile_size * i) + threadId];

            // S = QK^T, row_m = rowmax(S)
            float row_m = -INFINITY;
            for (int y = 0; y < col_tile_size; y++) {
                float sum = 0;
                for (int x = 0; x < d; x++) {
                    sum += Qi[(threadId * d) + x] * Kj[(y * d) + x];
                }
                sum *= scale;
                S[(col_tile_size * threadId) + y] = sum;
                row_m = max(row_m, sum);
            }

            float row_l = 0;
            for (int y = 0; y < col_tile_size; y++) {
                S[(col_tile_size * threadId) + y] = __expf(S[(col_tile_size * threadId) + y] - row_m);
                row_l += S[(col_tile_size * threadId) + y];
            }

            float row_m_new = max(row_m_prev, row_m);
            float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + 
                             (__expf(row_m - row_m_new) * row_l);

            // Update O, l, m
            for (int x = 0; x < d; x++) {
                float pv = 0;
                for (int y = 0; y < col_tile_size; y++) {
                    pv += S[(col_tile_size * threadId) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * i) + (threadId * d) + x] = 
                    (1 / row_l_new) * (
                        (row_l_prev * __expf(row_m_prev - row_m_new) * 
                         O[qkv_offset + (tile_size * i) + (threadId * d) + x]) +
                        (__expf(row_m - row_m_new) * pv)
                    );
            }
            m[l_m_offset + (row_tile_size * i) + threadId] = row_m_new;
            l[l_m_offset + (row_tile_size * i) + threadId] = row_l_new;
        }
        __syncthreads();
    }
}

void launch_flash_attention(
    const float* Q, const float* K, const float* V, float* O,
    const int B, const int H, const int N, const int D
) {
    const int col_tile_size = 32;
    const int row_tile_size = 32;
    
    const int num_col_tiles = (N + col_tile_size - 1) / col_tile_size; 
    const int num_row_tiles = (N + row_tile_size - 1) / row_tile_size;
    const float scale = 1.0f / sqrt(D);
    
    // Allocate and initialize running statistics
    float *d_l, *d_m;
    CUDA_CHECK(hipMalloc(&d_l, B * H * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_m, B * H * N * sizeof(float)));
    
    CUDA_CHECK(hipMemset(d_l, 0, B * H * N * sizeof(float)));
    float neg_inf = -INFINITY;
    CUDA_CHECK(hipMemset(d_m, neg_inf, B * H * N * sizeof(float)));
    
    // Calculate shared memory size
    const int shmem_size = (3 * col_tile_size * D * sizeof(float)) + (col_tile_size * row_tile_size * sizeof(float));
    int max_shmem_size;
    hipDeviceGetAttribute(&max_shmem_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    // printf("Max shared memory: %d, requested shared memory: %d\n", max_shmem_size, shmem_size);
    
    dim3 grid(B, H); 
    dim3 block(col_tile_size); 
    
    flash_attention_kernel<<<grid, block, shmem_size>>>(
        Q, K, V, N, D, num_col_tiles, num_row_tiles, col_tile_size, row_tile_size, scale, d_l, d_m, O
    );
    
    hipFree(d_l);
    hipFree(d_m);
}

BenchmarkResults run_flash_attention_benchmark(
    float* d_Q, float* d_K, float* d_V, float* d_output,
    int B, int N, int H, int d,
    int num_iterations) {
    
    BenchmarkResults results;
    results.individual_times.reserve(num_iterations);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    for (int i = 0; i < num_iterations; ++i) {
        // Zero out output buffer before each iteration
        CUDA_CHECK(hipMemset(d_output, 0, B * H * N * d * sizeof(float)));
        
        CUDA_CHECK(hipEventRecord(start));
        
        launch_flash_attention(d_Q, d_K, d_V, d_output, B, H, N, d);
        
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));

        float time_ms;
        CUDA_CHECK(hipEventElapsedTime(&time_ms, start, stop));
        results.individual_times.push_back(time_ms);
    }

    // Calculate statistics
    results.min_time_ms = *std::min_element(results.individual_times.begin(), results.individual_times.end());
    results.max_time_ms = *std::max_element(results.individual_times.begin(), results.individual_times.end());
    results.avg_time_ms = std::accumulate(results.individual_times.begin(), results.individual_times.end(), 0.0) / num_iterations;

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return results;
}

void print_benchmark_results(const std::string& name, const BenchmarkResults& results) {
    std::cout << "\n=== " << name << " Performance ===" << std::endl;
    std::cout << std::fixed << std::setprecision(3);
    std::cout << "Average time: " << results.avg_time_ms << " ms" << std::endl;
    std::cout << "Min time:     " << results.min_time_ms << " ms" << std::endl;
    std::cout << "Max time:     " << results.max_time_ms << " ms" << std::endl;
    std::cout << "Variance:     " << std::fixed << std::setprecision(6)
              << std::accumulate(results.individual_times.begin(), results.individual_times.end(), 0.0,
                               [&](double acc, double x) {
                                   return acc + (x - results.avg_time_ms) * (x - results.avg_time_ms);
                               }) / results.individual_times.size() << std::endl;
}

void initialize_cuda_device() {
    // Initialize CUDA device
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found!" << std::endl;
        exit(-1);
    }

    bool print_available_devices = false;
    if (print_available_devices) {
        // Print available devices
        std::cout << "Available CUDA devices:" << std::endl;
        for (int i = 0; i < deviceCount; i++) {
            hipDeviceProp_t prop;
            CUDA_CHECK(hipGetDeviceProperties(&prop, i));
            std::cout << "Device " << i << ": " << prop.name 
                    << " (Compute " << prop.major << "." << prop.minor << ")" 
                    << "\n\tGlobal Memory: " << prop.totalGlobalMem / (1024*1024) << " MB"
                    << "\n\tSMs: " << prop.multiProcessorCount
                    << "\n\tMax threads per block: " << prop.maxThreadsPerBlock
                    << "\n\tMax threads per SM: " << prop.maxThreadsPerMultiProcessor
                    << std::endl;
        }
    }

    // Select first device by default
    CUDA_CHECK(hipSetDevice(0));
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    std::cout << "\nUsing device 0: " << prop.name << std::endl;
}

int main() {
    initialize_cuda_device();
    const int B = 4;    // Batch size
    const int N = 64;   // Sequence length
    const int H = 8;    // Number of attention heads
    const int d = 64;   // Dimension per head
    const int num_iterations = 10;  // Number of benchmark iterations

    const int QKV_size = B * N * H * d;
    const int output_size = QKV_size;

    // Host tensors
    std::vector<float> h_Q(QKV_size), h_K(QKV_size), h_V(QKV_size), h_V_col(QKV_size);
    std::vector<float> h_output_cpu(output_size), h_output_gpu(output_size);

    initialize_tensor(h_Q, QKV_size);
    initialize_tensor(h_K, QKV_size);
    initialize_tensor(h_V, QKV_size);

    // Set V column major
    for (int b = 0; b < B; ++b) {
        for (int h = 0; h < H; ++h) {
            for (int i = 0; i < N; ++i) {
                for (int j = 0; j < d; ++j) {
                    h_V_col[b * H * N * d + h * N * d + j * N + i] = h_V[b * H * N * d + h * N * d + i * d + j];
                }
            }
        }
    }

    // Device tensors
    float *d_Q, *d_K, *d_V, *d_output, *d_V_col;
    CUDA_CHECK(hipMalloc(&d_Q, QKV_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_K, QKV_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_V, QKV_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, output_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_V_col, QKV_size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_Q, h_Q.data(), QKV_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_K, h_K.data(), QKV_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_V, h_V.data(), QKV_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_V_col, h_V_col.data(), QKV_size * sizeof(float), hipMemcpyHostToDevice));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Warmup runs
    flash_attention_cpu(h_Q.data(), h_K.data(), h_V.data(), h_output_cpu.data(), B, N, H, d);
    hipDeviceSynchronize();

    // Run benchmarks
    std::cout << "\nRunning benchmarks with " << num_iterations << " iterations..." << std::endl;
    std::cout << "Configuration: B=" << B << ", N=" << N << ", H=" << H << ", d=" << d << std::endl;

    auto cpu_results = run_cpu_benchmark(h_Q, h_K, h_V, h_output_cpu, B, N, H, d, num_iterations);

    auto gpu_results = run_gpu_benchmark(handle, d_Q, d_K, d_V_col, d_output, B, N, H, d, num_iterations);
    CUDA_CHECK(hipMemcpy(h_output_gpu.data(), d_output, output_size * sizeof(float), hipMemcpyDeviceToHost));

    // Column major to row major
    std::vector<float> h_output_gpu_row_major(output_size);
    for (int b = 0; b < B; ++b) {
        for (int h = 0; h < H; ++h) {
            for (int i = 0; i < N; ++i) {
                for (int j = 0; j < d; ++j) {
                    h_output_gpu_row_major[b * H * N * d + h * N * d + i * d + j] = h_output_gpu[b * H * N * d + h * N * d + j * N + i];
                }
            }
        }
    }

    auto flash_results = run_flash_attention_benchmark(d_Q, d_K, d_V, d_output, B, N, H, d, num_iterations);
    CUDA_CHECK(hipMemcpy(h_output_gpu.data(), d_output, output_size * sizeof(float), hipMemcpyDeviceToHost));

    // Print results
    print_benchmark_results("CPU", cpu_results);
    print_benchmark_results("GPU", gpu_results);
    print_benchmark_results("Flash", flash_results);
    

    // Verify results
    std::cout << "\nVerifying results..." << std::endl;
    bool passed = true;
    const float epsilon = 1e-3f;
    for (int i = 0; i < output_size; ++i) {
        //std::cout << h_output_cpu[i] << " " << h_output_gpu[i] << std::endl;
        if (fabs(h_output_cpu[i] - h_output_gpu_row_major[i]) > epsilon) {
            std::cout << "Mismatch at index " << i << ": CPU = " << h_output_cpu[i]
                     << ", GPU = " << h_output_gpu_row_major[i] << std::endl;
            passed = false;
            break;
        }
    }

    for (int i = 0; i < output_size; ++i) {
        if (fabs(h_output_cpu[i] - h_output_gpu[i]) > epsilon) {
            std::cout << "Mismatch at index " << i << ": CPU = " << h_output_cpu[i]
                     << ", Flash Attention = " << h_output_gpu[i] << std::endl;
            passed = false;
            break;
        }
    }

    if (passed) {
        std::cout << "All results match within epsilon = " << epsilon << std::endl;
    } else {
        std::cout << "Results don't match!" << std::endl;
    }

    // Print speedup
    double speedup = cpu_results.min_time_ms / gpu_results.min_time_ms;
    std::cout << "\nGPU Speedup: " << std::fixed << std::setprecision(2) << speedup << "x" << std::endl;

    double flash_speedup = gpu_results.min_time_ms / flash_results.min_time_ms;
    std::cout << "Flash Speedup: " << std::fixed << std::setprecision(2) << flash_speedup << "x" << std::endl;

    // Cleanup
    CUBLAS_CHECK(hipblasDestroy(handle));
    CUDA_CHECK(hipFree(d_Q));
    CUDA_CHECK(hipFree(d_K));
    CUDA_CHECK(hipFree(d_V));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_V_col));

    return 0;
}
