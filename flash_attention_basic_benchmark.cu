#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <cfloat>
#include <cassert>
#include <chrono>
#include <iomanip>
#include <algorithm>  // for min_element, max_element
#include <numeric>    // for accumulate
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "flash_attention_basic.cu"
#include "utils.cuh"

void flash_attention_cpu(const float* Q, const float* K, const float* V, float* output, int B, int N, int H, int d) {
    std::vector<float> scores(N * N);
    bool debug = false;

    for (int b = 0; b < B; ++b) {
        for (int h = 0; h < H; ++h) {
            const float* q = Q + b * H * N * d + h * N * d;
            const float* k = K + b * H * N * d + h * N * d;
            const float* v = V + b * H * N * d + h * N * d;
            float* o = output + b * H * N * d + h * N * d;

            // Compute Q * K^T (row-major)
            for (int i = 0; i < N; ++i) {
                for (int j = 0; j < N; ++j) {
                    float sum = 0.0f;
                    for (int d_i = 0; d_i < d; ++d_i) {
                        // Q[i,d_i] * K[j,d_i] for row-major layout
                        sum += q[i * d + d_i] * k[j * d + d_i];
                    }
                    scores[i * N + j] = sum / sqrtf(d);
                    
                }
            }

            if (debug) {
                //print scores
                std::cout << "CPU Scores: ";
                for (int i = 0; i < N * N; ++i) {
                    std::cout << scores[i] << " ";
                }
                std::cout << std::endl;
            }

            // Apply softmax row by row
            for (int i = 0; i < N; ++i) {
                // first find max for numerical stability
                float max_val = -std::numeric_limits<float>::infinity();
                for (int j = 0; j < N; ++j) {
                    max_val = std::max(max_val, scores[i * N + j]);
                }

                // compute exp and sum
                float sum_exp = 0.0f;
                for (int j = 0; j < N; ++j) {
                    scores[i * N + j] = expf(scores[i * N + j] - max_val);
                    sum_exp += scores[i * N + j];
                }

                // normalize
                for (int j = 0; j < N; ++j) {
                    scores[i * N + j] /= sum_exp;
                }
            }

            if (debug) {
                // print cpu softmax
                std::cout << "CPU Softmax: ";
                for (int i = 0; i < N * N; ++i) {
                    std::cout << scores[i] << " ";
                }
                std::cout << std::endl;
            }

            // Compute scores * V (row-major)
            for (int i = 0; i < N; ++i) {
                for (int d_i = 0; d_i < d; ++d_i) {
                    float sum = 0.0f;
                    for (int j = 0; j < N; ++j) {
                        // scores[i,j] * V[j,d_i] for row-major layout
                        sum += scores[i * N + j] * v[j * d + d_i];
                    }
                    o[i * d + d_i] = sum;
                }
            }
        }
    }
}

void initialize_tensor(std::vector<float>& tensor, int size) {
    for (int i = 0; i < size; ++i) {
        tensor[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

struct BenchmarkResults {
    double avg_time_ms;
    double min_time_ms;
    double max_time_ms;
    std::vector<double> individual_times;
};

BenchmarkResults run_cpu_benchmark(const std::vector<float>& h_Q, 
                                 const std::vector<float>& h_K,
                                 const std::vector<float>& h_V,
                                 std::vector<float>& h_output,
                                 int B, int N, int H, int d,
                                 int num_iterations) {
    BenchmarkResults results;
    results.individual_times.reserve(num_iterations);

    for (int i = 0; i < num_iterations; ++i) {
        auto start = std::chrono::high_resolution_clock::now();
        
        flash_attention_cpu(h_Q.data(), h_K.data(), h_V.data(), h_output.data(), B, N, H, d);
        
        auto end = std::chrono::high_resolution_clock::now();
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count();
        results.individual_times.push_back(time_ms);
    }

    // Calculate statistics
    results.min_time_ms = *std::min_element(results.individual_times.begin(), results.individual_times.end());
    results.max_time_ms = *std::max_element(results.individual_times.begin(), results.individual_times.end());
    results.avg_time_ms = std::accumulate(results.individual_times.begin(), results.individual_times.end(), 0.0) / num_iterations;

    return results;
}

__global__ void columnwise_softmax(float* input, float* output, int N) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread handles one column
    if (tid < N) {
        // Find max element in the column for numerical stability
        float max_val = input[tid];
        for (int i = 1; i < N; i++) {
            max_val = fmaxf(max_val, input[i * N + tid]);
        }
        
        // Calculate exp(x - max) and sum
        float sum = 0.0f;
        for (int i = 0; i < N; i++) {
            float val = expf(input[i * N + tid] - max_val);
            output[i * N + tid] = val;
            sum += val;
        }
        
        // Normalize by sum
        for (int i = 0; i < N; i++) {
            output[i * N + tid] /= sum;
        }
    }
}

// V is specifically in column-major format
BenchmarkResults run_gpu_benchmark(hipblasHandle_t handle,
                                 float* d_Q, float* d_K, float* d_V, float* d_output,
                                 int B, int N, int H, int d,
                                 int num_iterations) {
    BenchmarkResults results;
    bool debug = false;
    results.individual_times.reserve(num_iterations);

    CUBLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
    CUBLAS_CHECK(hipblasSetStream(handle, 0));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Calculate leading dimensions and strides
    int ldq = d;      
    int ldk = d;      
    int lds = N;    

    for (int i = 0; i < num_iterations; ++i) {
        CUDA_CHECK(hipEventRecord(start));

        float alpha_qk = 1.0f / sqrtf(d);
        float alpha_v = 1.0f;
        float beta = 0.0f;

        float* d_scores;
        CUDA_CHECK(hipMalloc(&d_scores, B * H * N * N * sizeof(float)));

        // Process each batch and head
        for (int b = 0; b < B; ++b) {
            for (int h = 0; h < H; ++h) {
                float* current_Q = d_Q + (b * H * N * d) + (h * N * d);
                float* current_K = d_K + (b * H * N * d) + (h * N * d);
                float* current_V = d_V + (b * H * N * d) + (h * N * d);
                float* current_scores = d_scores + (b * H * N * N) + (h * N * N);
                float* current_output = d_output + (b * H * N * d) + (h * N * d);

                // Q * K^T computation
                CUBLAS_CHECK(hipblasSgemm(handle,
                    HIPBLAS_OP_T,  
                    HIPBLAS_OP_N, 
                    N, N, d,        // m, n, k dimensions
                    &alpha_qk,      // scaling factor
                    current_Q, ldq,  // Q matrix, N x d
                    current_K, ldk,  // K matrix, N x d
                    &beta,   
                    current_scores, lds)); // output scores which is N x N

                if (debug) {
                    // print current scores
                    float* h_scores = new float[N * N];
                    CUDA_CHECK(hipMemcpy(h_scores, current_scores, N * N * sizeof(float), hipMemcpyDeviceToHost));
                    std::cout << "Scores: ";
                    for (int i = 0; i < N * N; ++i) {
                        std::cout << h_scores[i] << " ";
                    }
                    std::cout << std::endl;
                }

                int threadsPerBlock = std::min(N, 1024);
                int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
                columnwise_softmax<<<blocksPerGrid, threadsPerBlock>>>(current_scores, current_scores, N);
                CUDA_CHECK(hipGetLastError());

                if (debug) {
                    // print gpu softmax
                    float* h_scores_softmax = new float[N * N];
                    CUDA_CHECK(hipMemcpy(h_scores_softmax, current_scores, N * N * sizeof(float), hipMemcpyDeviceToHost));
                    std::cout << "GPU Softmax: ";
                    for (int i = 0; i < N * N; ++i) {
                        std::cout << h_scores_softmax[i] << " ";
                    }
                    std::cout << std::endl;
                }
                
                CUBLAS_CHECK(hipblasSgemm(handle,
                    HIPBLAS_OP_N,  
                    HIPBLAS_OP_N, 
                    N, d, N,        // m, n, k dimensions
                    &alpha_v,       // same scale
                    current_scores, lds, // scores matrix which are N x N
                    current_V, N,     // V matrix is N x d
                    &beta,    
                    current_output, N)); // output matrix is N x d
                
            }
        }

        CUDA_CHECK(hipFree(d_scores));
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));

        float time_ms;
        CUDA_CHECK(hipEventElapsedTime(&time_ms, start, stop));
        results.individual_times.push_back(time_ms);
    }

    std::sort(results.individual_times.begin(), results.individual_times.end());
    results.min_time_ms = results.individual_times.front();
    results.max_time_ms = results.individual_times.back();
    results.avg_time_ms = std::accumulate(results.individual_times.begin(), results.individual_times.end(), 0.0) / num_iterations;
    //results.median_time_ms = results.individual_times[num_iterations / 2];

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return results;
}

BenchmarkResults run_flash_attention_benchmark(
    float* d_Q, float* d_K, float* d_V, float* d_output,
    int B, int N, int H, int d,
    int num_iterations,
    void (*launch_flash_attention_fn)(const float*, const float*, const float*, float*, const int, const int, const int, const int)) {
    
    BenchmarkResults results;
    results.individual_times.reserve(num_iterations);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    for (int i = 0; i < num_iterations; ++i) {
        // Zero out output buffer before each iteration
        CUDA_CHECK(hipMemset(d_output, 0, B * H * N * d * sizeof(float)));
        
        CUDA_CHECK(hipEventRecord(start));
        
        launch_flash_attention_fn(d_Q, d_K, d_V, d_output, B, H, N, d);
        
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));

        float time_ms;
        CUDA_CHECK(hipEventElapsedTime(&time_ms, start, stop));
        results.individual_times.push_back(time_ms);
    }

    // Calculate statistics
    results.min_time_ms = *std::min_element(results.individual_times.begin(), results.individual_times.end());
    results.max_time_ms = *std::max_element(results.individual_times.begin(), results.individual_times.end());
    results.avg_time_ms = std::accumulate(results.individual_times.begin(), results.individual_times.end(), 0.0) / num_iterations;

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return results;
}

void print_benchmark_results(const std::string& name, const BenchmarkResults& results) {
    std::cout << "\n=== " << name << " Performance ===" << std::endl;
    std::cout << std::fixed << std::setprecision(3);
    std::cout << "Average time: " << results.avg_time_ms << " ms" << std::endl;
    std::cout << "Min time:     " << results.min_time_ms << " ms" << std::endl;
    std::cout << "Max time:     " << results.max_time_ms << " ms" << std::endl;
    std::cout << "Variance:     " << std::fixed << std::setprecision(6)
              << std::accumulate(results.individual_times.begin(), results.individual_times.end(), 0.0,
                               [&](double acc, double x) {
                                   return acc + (x - results.avg_time_ms) * (x - results.avg_time_ms);
                               }) / results.individual_times.size() << std::endl;
}

void initialize_cuda_device() {
    // Initialize CUDA device
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found!" << std::endl;
        exit(-1);
    }

    bool print_available_devices = false;
    if (print_available_devices) {
        // Print available devices
        std::cout << "Available CUDA devices:" << std::endl;
        for (int i = 0; i < deviceCount; i++) {
            hipDeviceProp_t prop;
            CUDA_CHECK(hipGetDeviceProperties(&prop, i));
            std::cout << "Device " << i << ": " << prop.name 
                    << " (Compute " << prop.major << "." << prop.minor << ")" 
                    << "\n\tGlobal Memory: " << prop.totalGlobalMem / (1024*1024) << " MB"
                    << "\n\tSMs: " << prop.multiProcessorCount
                    << "\n\tMax threads per block: " << prop.maxThreadsPerBlock
                    << "\n\tMax threads per SM: " << prop.maxThreadsPerMultiProcessor
                    << std::endl;
        }
    }

    // Select first device by default
    CUDA_CHECK(hipSetDevice(0));
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    std::cout << "\nUsing device 0: " << prop.name << std::endl;
}

// Returns RRMSE
float compare_results(std::vector<float>& h1, std::vector<float>& h2, int32_t output_size) {
    double mse = 0.0;
    double ref_mean_square = 0.0;
    for (int32_t i = 0; i < output_size; ++i) {
        float diff = h2[i] - h1[i];
        mse += diff * diff;
        ref_mean_square += h1[i] * h1[i];
    }
    mse /= output_size;
    ref_mean_square /= output_size;
    float rmse = std::sqrt(mse);
    float rel_rmse = rmse / std::sqrt(ref_mean_square);
    return rel_rmse;
}

int main() {
    srand(0);
    initialize_cuda_device();
    bool run_cpu = true;

    const int B = 32;    // Batch size
    const int N = 64;   // Sequence length
    const int H = 8;    // Number of attention heads
    const int d = 64;   // Dimension per head
    const int num_iterations = 10;  // Number of benchmark iterations

    const int QKV_size = B * N * H * d;
    const int output_size = QKV_size;

    // Host tensors
    std::vector<float> h_Q(QKV_size), h_K(QKV_size), h_V(QKV_size), h_V_col(QKV_size);
    std::vector<float> h_output_cpu(output_size), h_output_gpu(output_size), h_output_gpu_base(output_size), h_output_flash(output_size);

    initialize_tensor(h_Q, QKV_size);
    initialize_tensor(h_K, QKV_size);
    initialize_tensor(h_V, QKV_size);

    // Set V column major
    for (int b = 0; b < B; ++b) {
        for (int h = 0; h < H; ++h) {
            for (int i = 0; i < N; ++i) {
                for (int j = 0; j < d; ++j) {
                    h_V_col[b * H * N * d + h * N * d + j * N + i] = h_V[b * H * N * d + h * N * d + i * d + j];
                }
            }
        }
    }

    // Device tensors
    float *d_Q, *d_K, *d_V, *d_output, *d_V_col;
    CUDA_CHECK(hipMalloc(&d_Q, QKV_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_K, QKV_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_V, QKV_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, output_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_V_col, QKV_size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_Q, h_Q.data(), QKV_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_K, h_K.data(), QKV_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_V, h_V.data(), QKV_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_V_col, h_V_col.data(), QKV_size * sizeof(float), hipMemcpyHostToDevice));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // Warmup runs
    flash_attention_cpu(h_Q.data(), h_K.data(), h_V.data(), h_output_cpu.data(), B, N, H, d);
    hipDeviceSynchronize();

    // Run benchmarks
    std::cout << "\nRunning benchmarks with " << num_iterations << " iterations..." << std::endl;
    std::cout << "Configuration: B=" << B << ", N=" << N << ", H=" << H << ", d=" << d << std::endl;

    BenchmarkResults cpu_results;
    if (run_cpu) {
        cpu_results = run_cpu_benchmark(h_Q, h_K, h_V, h_output_cpu, B, N, H, d, num_iterations);
    }

    BenchmarkResults gpu_results;
    gpu_results = run_gpu_benchmark(handle, d_Q, d_K, d_V_col, d_output, B, N, H, d, num_iterations);
    CUDA_CHECK(hipMemcpy(h_output_gpu.data(), d_output, output_size * sizeof(float), hipMemcpyDeviceToHost));

    // Column major to row major
    for (int b = 0; b < B; ++b) {
        for (int h = 0; h < H; ++h) {
            for (int i = 0; i < N; ++i) {
                for (int j = 0; j < d; ++j) {
                    h_output_gpu_base[b * H * N * d + h * N * d + i * d + j] = h_output_gpu[b * H * N * d + h * N * d + j * N + i];
                }
            }
        }
    }

    auto flash_results = run_flash_attention_benchmark(d_Q, d_K, d_V, d_output, B, N, H, d, num_iterations, launch_flash_attention_basic);
    CUDA_CHECK(hipMemcpy(h_output_flash.data(), d_output, output_size * sizeof(float), hipMemcpyDeviceToHost));

    // Print results
    if (run_cpu) {
        print_benchmark_results("CPU", cpu_results);
    }
    print_benchmark_results("GPU", gpu_results);
    print_benchmark_results("Flash", flash_results);
    
    // Verify results
    std::cout << "\nVerifying results..." << std::endl;
    if (run_cpu) {
        float rrmse_cpu_gpubase = compare_results(h_output_cpu, h_output_gpu_base, output_size);
        float rrmse_cpu_flash = compare_results(h_output_cpu, h_output_flash, output_size);
        std::cout << "RRMSE CPU vs GPU Base: " << rrmse_cpu_gpubase << std::endl;
        std::cout << "RRMSE CPU vs Flash:    " << rrmse_cpu_flash << std::endl;
    } else {
        float rrmse_gpu_flash = compare_results(h_output_gpu_base, h_output_flash, output_size);
        std::cout << "RRMSE GPU Base vs Flash: " << rrmse_gpu_flash << std::endl;
    }

    if (run_cpu) {
        double speedup = cpu_results.min_time_ms / gpu_results.min_time_ms;
        std::cout << "\nGPU Speedup: " << std::fixed << std::setprecision(2) << speedup << "x" << std::endl;
    }
    double flash_speedup = gpu_results.min_time_ms / flash_results.min_time_ms;
    std::cout << "Flash Speedup: " << std::fixed << std::setprecision(2) << flash_speedup << "x" << std::endl;

    CUBLAS_CHECK(hipblasDestroy(handle));
    CUDA_CHECK(hipFree(d_Q));
    CUDA_CHECK(hipFree(d_K));
    CUDA_CHECK(hipFree(d_V));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_V_col));

    return 0;
}